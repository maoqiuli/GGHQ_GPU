/* Copyright 2019 ComputerGraphics Tuebingen. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License.
==============================================================================*/
// Authors: Fabian Groh, Lukas Ruppert, Patrick Wieschollek, Hendrik P.A. Lensch
//
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <gflags/gflags.h>
#include <glog/logging.h>
#include <stdio.h>

#include <hipcub/hipcub.hpp>
#include <iostream>
#include <vector>

#include "ggnn/cuda_knn_ggnn.cuh"
#include "ggnn/utils/cuda_knn_constants.cuh"
#include "ggnn/utils/cuda_knn_dataset.cuh"
#include "ggnn/utils/cuda_knn_utils.cuh"

DEFINE_string(base_filename, "", "path to file with base vectors");
DEFINE_string(query_filename, "", "path to file with perform_query vectors");
DEFINE_string(groundtruth_filename, "",
              "path to file with groundtruth vectors");
DEFINE_string(graph_filename, "",
              "path to file that contains the serialized graph");
DEFINE_double(tau, 0.5, "Parameter tau");
DEFINE_int32(refinement_iterations, 2, "Number of refinement iterations");
DEFINE_int32(gpu_id, 0, "GPU id");
DEFINE_bool(grid_search, false,
            "Perform queries for a wide range of parameters.");

int main(int argc, char* argv[]) {
  google::InitGoogleLogging(argv[0]);
  google::LogToStderr();

  gflags::SetUsageMessage(
      "GGNN: Graph-based GPU Nearest Neighbor Search\n"
      "by Fabian Groh, Lukas Ruppert, Patrick Wieschollek, Hendrik P.A. "
      "Lensch\n"
      "(c) 2020 Computer Graphics University of Tuebingen");
  gflags::SetVersionString("1.0.0");
  google::ParseCommandLineFlags(&argc, &argv, true);

  LOG(INFO) << "Reading files";
  CHECK(file_exists(FLAGS_base_filename))
      << "File for base vectors has to exist";
  CHECK(file_exists(FLAGS_query_filename))
      << "File for perform_query vectors has to exist";

  CHECK_GE(FLAGS_tau, 0) << "Tau has to be bigger or equal 0.";
  CHECK_GE(FLAGS_refinement_iterations, 0)
      << "The number of refinement iterations has to be non-negative.";

  // ####################################################################
  // compile-time configuration
  //
  // data types
  //
  /// data type for addressing points (needs to be able to represent N)
  using KeyT = int32_t;
  /// data type of the dataset (e.g., char, int, float)
  using BaseT = float;
  /// data type of computed distances
  using ValueT = float;
  /// data type for addressing base-vectors (needs to be able to represent N*D)
  using BAddrT = uint32_t;
  /// data type for addressing the graph (needs to be able to represent
  /// N*KBuild)
  using GAddrT = uint32_t;
  //
  // dataset configuration (here: SIFT1M)
  //
  /// dimension of the dataset
  const int D = 128;
  /// distance measure (Euclidean or Cosine)
  const DistanceMeasure measure = Euclidean;
  //
  // search-graph configuration
  //
  /// number of neighbors per point in the graph
  const int KBuild = 24;
  /// maximum number of inverse/symmetric links (KBuild / 2 usually works best)
  const int KF = KBuild / 2;
  /// segment/batch size (needs to be > KBuild-KF)
  const int S = 32;
  /// graph height / number of layers (4 usually performs best)
  const int L = 4;
  //
  // query configuration
  //
  /// number of neighbors to search for
  const int KQuery = 10;

  static_assert(KBuild - KF < S,
                "there are not enough points to fill the local neighbor list!");

  LOG(INFO) << "Using the following parameters " << KBuild << " (KBuild) " << KF
            << " (KF) " << S << " (S) " << L << " (L) " << D << " (D) ";

  // Set the requested GPU id, if possible.
  {
    int numGpus;
    hipGetDeviceCount(&numGpus);
    CHECK_GE(FLAGS_gpu_id, 0) << "This GPU does not exist";
    CHECK_LT(FLAGS_gpu_id, numGpus) << "This GPU does not exist";

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, FLAGS_gpu_id);
    LOG(INFO) << "device name: " << prop.name;
  }
  hipSetDevice(FLAGS_gpu_id);

  typedef GGNN<measure, KeyT, ValueT, GAddrT, BaseT, BAddrT, D, KBuild, KF,
               KQuery, S>
      GGNN;
  GGNN m_ggnn{FLAGS_base_filename, FLAGS_query_filename,
              FLAGS_groundtruth_filename, L, static_cast<float>(FLAGS_tau)};

  m_ggnn.ggnnMain(FLAGS_graph_filename, FLAGS_refinement_iterations);

  auto query_function = [&m_ggnn](const float tau_query) {
    hipMemcpyToSymbol(HIP_SYMBOL(c_tau_query), &tau_query, sizeof(float));
    LOG(INFO) << "--";
    LOG(INFO) << "Query with tau_query " << tau_query;
    // faster for C@1 = 99%
    // LOG(INFO) << "fast query (good for C@1)";
    // m_ggnn.queryLayer<32, 200, 256, 64>();
    // better for C@10 > 99%
    // LOG(INFO) << "regular query (good for C@10)";
    m_ggnn.queryLayer<32, 400, 448, 64>();
    // expensive, can get to 99.99% C@10
    // m_ggnn.queryLayer<128, 2000, 2048, 256>();
  };

  const int N_query_total = m_ggnn.dataset.N_query;

  LOG(INFO) << "--";
  LOG(INFO) << "99% R@1 (using -tau 0.5 -refinement_iterations 2):";

  m_ggnn.dataset.template checkForDuplicatesInGroundTruth<measure, ValueT>(
      KQuery);

  if (FLAGS_grid_search) {
    LOG(INFO) << "--";
    LOG(INFO) << "grid-search:";
    for (int i = 1; i <= N_query_total; i += 500) {
      m_ggnn.dataset.N_query = i;
      query_function(0.51f);

      if (i == 1) i = 0;
    }
  } else {  // by default, just execute a few queries
    LOG(INFO) << "--";
    LOG(INFO) << "1, 10, 100, 1000, 10000:";
    for (int i = 1; i <= N_query_total; i *= 10) {
      m_ggnn.dataset.N_query = i;
      query_function(0.51f);
    }
  }

  printf("done! \n");
  gflags::ShutDownCommandLineFlags();
  return 0;
}
